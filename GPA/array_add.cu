#include <stdio.h>
#include <hip/hip_runtime.h>

// Error checking macro
#define cudaCheckError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// Kernel for vector addition
__global__ void vectorAdd(int *A, int *B, int *C, int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    int N = 10;
    int size = N * sizeof(int);

    int h_A[N], h_B[N], h_C[N];
    for (int i = 0; i < N; i++) {
        h_A[i] = i;
        h_B[i] = i * 2;
    }

    int *d_A, *d_B, *d_C;
    cudaCheckError(hipMalloc((void**)&d_A, size));
    cudaCheckError(hipMalloc((void**)&d_B, size));
    cudaCheckError(hipMalloc((void**)&d_C, size));

    cudaCheckError(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    cudaCheckError(hipGetLastError());
    cudaCheckError(hipDeviceSynchronize());

    cudaCheckError(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));

    printf("Result:\n");
    for (int i = 0; i < N; i++) {
        printf("%d + %d = %d\n", h_A[i], h_B[i], h_C[i]);
    }

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    return 0;
}